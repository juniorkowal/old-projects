#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <bits/stdc++.h>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

void dodawanie_macierzy_cpu(double ** M, double ** N, double ** W, int size) {
  int i, j;

  for (i = 0; i < size; i++) {
    for (j = 0; j < size; j++) {
      W[i][j] = M[i][j] + N[i][j];
    }
  }
}

void mnozenie_macierzy_cpu(double ** M, double ** N, double ** W, int size) {
  int i, j, k, a, b = 1;

  for (i = 0; i < size; i++) {
    a = 1;
    b--;
    for (j = 0; j < size; j++) {
      a--;
      for (k = 0; k < size; k++) {
        W[i][j] += M[i][j + a + k] * N[i + b + k][j];
      }
    }
  }
}

void mnozenie_liczba_cpu(double ** M, int liczba, double ** W, int size) {
  int i, j;

  for (i = 0; i < size; i++) {
    for (j = 0; j < size; j++) {
      W[i][j] = M[i][j] * liczba;
    }
  }
}

void transpose_cpu(double ** M, double ** W, int size) {
  int i, j;

  for (i = 0; i < size; i++) {
    for (j = 0; j < size; j++) {
      W[i][j] = M[j][i];
    }
  }
}

__global__ void mnozenie_gpu(double * M, double * N, double * W, int size) {
  int wiersz = blockIdx.y * blockDim.y + threadIdx.y;
  int kolumna = blockIdx.x * blockDim.x + threadIdx.x;
  double tmpSum = 0;

  for (int i = 0; i < size; i++) {
    tmpSum += M[wiersz * size + i] * N[i * size + kolumna];
  }
  W[wiersz * size + kolumna] = tmpSum;
  __syncthreads();
}

__global__ void dodawanie_gpu(double * M, double * N, double * W, int size) {
  int wiersz = blockIdx.y * blockDim.y + threadIdx.y;
  int kolumna = blockIdx.x * blockDim.x + threadIdx.x;

  W[wiersz * size + kolumna] = M[wiersz * size + kolumna] + N[wiersz * size + kolumna];
  __syncthreads();
}

__global__ void transpose_gpu(double * M, double * W, int size) {
  int wiersz = blockIdx.y * blockDim.y + threadIdx.y;
  int kolumna = blockIdx.x * blockDim.x + threadIdx.x;

  W[wiersz * size + kolumna] = M[kolumna * size + wiersz];
  __syncthreads();
}

__global__ void mnozenie_przez_liczbe_gpu_kernel(double * M, int liczba, double * W, int size) {
  int wiersz = blockIdx.y * blockDim.y + threadIdx.y;
  int kolumna = blockIdx.x * blockDim.x + threadIdx.x;

  W[wiersz * size + kolumna] = M[wiersz * size + kolumna] * liczba;
  __syncthreads();
}

void macierz_gpu(double * A, double * B, double * wynik, double * X, int u, int w, int size) {
  dim3 threadsPerBlock(size, size);
  dim3 blocksPerGrid(1, 1);
  if (size * size > 512) {
    threadsPerBlock.x = 512;
    threadsPerBlock.y = 512;
    blocksPerGrid.x = ceil(double(size) / double(threadsPerBlock.x));
    blocksPerGrid.y = ceil(double(size) / double(threadsPerBlock.y));
  }
  mnozenie_gpu << < blocksPerGrid, threadsPerBlock >>> (A, B, wynik, size);
  dodawanie_gpu << < blocksPerGrid, threadsPerBlock >>> (A, wynik, X, size);
  transpose_gpu << < blocksPerGrid, threadsPerBlock >>> (A, wynik, size);
  mnozenie_przez_liczbe_gpu_kernel << < blocksPerGrid, threadsPerBlock >>> (wynik, u, wynik, size);
  dodawanie_gpu << < blocksPerGrid, threadsPerBlock >>> (X, wynik, X, size);
  w = -w;
  mnozenie_przez_liczbe_gpu_kernel << < blocksPerGrid, threadsPerBlock >>> (B, w, wynik, size);
  dodawanie_gpu << < blocksPerGrid, threadsPerBlock >>> (X, wynik, X, size);
}

double blad_wynikow(double ** X_cpu, double * X_gpu, int size) {
  double tmpMaxCPU = 0;
  double tmpMaxDIFF = 0;
  double error = 0;

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      if ((X_cpu[i][j] - X_gpu[i * size + j]) > tmpMaxDIFF) {
        tmpMaxDIFF = X_cpu[i][j] - X_gpu[i * size + j];
      }
      if (X_cpu[i][j] > tmpMaxCPU) {
        tmpMaxCPU = X_cpu[i][j];
      }
    }
  }

  error = tmpMaxDIFF / tmpMaxCPU;
  return error;
}

int main() {
  srand(time(NULL));

  int czy_wyswietlac, size, u = 5, w = 0;

  printf("Podaj size macierzy:");
  scanf("%d", & size);
  printf("0 - nie wyswietlaj wynikow; 1 - wyswietlaj wyniki:");
  scanf("%d", & czy_wyswietlac);

  double ** wynik = new double * [size];
  double ** X = new double * [size];
  double ** A = new double * [size];
  double ** B = new double * [size];

  for (int i = 0; i < size; ++i) {
    wynik[i] = new double[size];
  }

  for (int i = 0; i < size; ++i) {
    X[i] = new double[size];
  }

  for (int i = 0; i < size; ++i) {
    A[i] = new double[size];
  }

  for (int i = 0; i < size; ++i) {
    B[i] = new double[size];
  }

  printf("Macierz A:\n");
  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      A[i][j] = rand() % 100;
      if (czy_wyswietlac == 1) {
        printf("%f\t", A[i][j]);
      }
    }
    if (czy_wyswietlac == 1) {
      printf("\n");
    }
  }
  if (czy_wyswietlac == 1) {
    printf("\n");
  }

  printf("Macierz B:\n");
  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      B[i][j] = rand() % 100;
      if (czy_wyswietlac == 1) {
        printf("%f\t", B[i][j]);
      }
    }
    if (czy_wyswietlac == 1) {
      printf("\n");
    }
  }
  if (czy_wyswietlac == 1) {
    printf("\n");
  }

  clock_t cpu_start, cpu_end;
  hipEvent_t gpu_start, gpu_end;
  hipEventCreate( & gpu_start);
  hipEventCreate( & gpu_end);

  cpu_start = clock();
  mnozenie_macierzy_cpu(A, B, X, size);
  dodawanie_macierzy_cpu(X, A, X, size);
  transpose_cpu(A, wynik, size);
  mnozenie_liczba_cpu(wynik, u, wynik, size);
  dodawanie_macierzy_cpu(wynik, X, X, size);
  w = -w;
  mnozenie_liczba_cpu(B, w, wynik, size);
  dodawanie_macierzy_cpu(wynik, X, X, size);
  cpu_end = clock();

  printf("Macierz X_cpu:\n");
  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      if (czy_wyswietlac == 1) {
        printf("%f\t", X[i][j]);
      }
    }
    if (czy_wyswietlac == 1) {
      printf("\n");
    }
  }
  if (czy_wyswietlac == 1) {
    printf("\n");
  }

  static
  const int n_el = size * size;
  static
  const size_t size = n_el * sizeof(double);

  double * h_A = (double * ) malloc(size);
  double * h_B = (double * ) malloc(size);
  double * h_C = (double * ) malloc(size);
  double * h_D = (double * ) malloc(size);

  double * d_A, * d_B, * d_C, * d_D;

  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      h_A[i * size + j] = A[i][j];
      h_B[i * size + j] = B[i][j];
    }
  }

  hipMalloc( & d_A, size);
  hipMalloc( & d_B, size);
  hipMalloc( & d_C, size);
  hipMalloc( & d_D, size);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  hipEventRecord(gpu_start);
  macierz_gpu(d_A, d_B, d_C, d_D, u, w, size);
  hipEventRecord(gpu_end);

  hipMemcpy(h_D, d_D, size, hipMemcpyDeviceToHost);

  printf("Macierz X_gpu:\n");
  for (int i = 0; i < size; i++) {
    for (int j = 0; j < size; j++) {
      if (czy_wyswietlac == 1) {
        printf("%f\t", h_D[i * size + j]);
      }
    }
    if (czy_wyswietlac == 1) {
      printf("\n");
    }
  }
  if (czy_wyswietlac == 1) {
    printf("\n");
  }

  double blad = 0;
  blad = blad_wynikow(X, h_D, size);
  printf("Bląd wyniku miedzy CPU a GPU wynosi: %f\t", blad);
  printf("\n");

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_D);

  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
  delete[] h_D;

  hipEventSynchronize(gpu_end);
  float czas = 0;
  hipEventElapsedTime( & czas, gpu_start, gpu_end);
  double czas_gpu = czas / 1000;
  cout << "Czas na GPU: " << czas_gpu << " sekund" << endl;
  float czas_cpu = float(cpu_end - cpu_start) / float(CLOCKS_PER_SEC);
  cout << "Czas CPU : " << czas_cpu << " sekund " << endl;
  float ile_razy_szybsze = czas_cpu / czas_gpu;
  cout << "GPU było " << ile_razy_szybsze << " razy szybsze niż CPU" << endl;
  return 0;
}