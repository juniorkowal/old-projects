#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>

using namespace std;

__global__ void sprawdzanie_gpu(char * d, unsigned long long int pierwiastek, unsigned long long int liczba) {
  unsigned long long int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i <= pierwiastek && liczba % (i * 2 + 3) == 0) {
    * d = 1;
  }
  __syncthreads();
}

bool pierwsza(long long int p) {
  bool nie_pierwsza = 0;
  if (p < 3) {
    nie_pierwsza = 0;
    return nie_pierwsza;
  }
  if (p % 2 == 0) {
    nie_pierwsza = 1;
    return nie_pierwsza;
  } else {
    for (int i = 3; i <= int(ceil(sqrt(p))); i += 2) {
      if (p % i == 0) {
        nie_pierwsza = 1;
        break;
      }

    }
    return nie_pierwsza;
  }
}

int main(void) {
  unsigned long long int liczba;
  bool czy_pierwsza;
  unsigned long long int pierwiastek;

  int watki;
  long long int bloki;

  clock_t start, end;

  hipEvent_t startGPU, stopGPU;
  hipEventCreate( & startGPU);
  hipEventCreate( & stopGPU);

  cout << "Podaj liczbe:" << endl;
  cin >> liczba;

  pierwiastek = int(ceil(sqrt(liczba)));

  size_t size = sizeof(char);
  char * wyjscie = (char * ) malloc(size);

  * wyjscie = 0;

  char * dev_a;
  hipMalloc( & dev_a, size);

  hipMemcpy(dev_a, wyjscie, size, hipMemcpyHostToDevice);

  if (pierwiastek <= 1024) {
    bloki = 1;
    watki = pierwiastek;
  } else {
    watki = 1024;
    bloki = int(ceil(pierwiastek / watki));
  }

  hipEventRecord(startGPU);
  sprawdzanie_gpu << < bloki, watki >>> (dev_a, pierwiastek, liczba);
  hipEventRecord(stopGPU);

  hipMemcpy(wyjscie, dev_a, size, hipMemcpyDeviceToHost);

  if (liczba <= 3) {
    * wyjscie = 0;
  }
  if (liczba % 2 == 0 && liczba != 2) {
    * wyjscie = 1;
  }

  hipEventSynchronize(stopGPU);
  float milliseconds = 0;
  hipEventElapsedTime( & milliseconds, startGPU, stopGPU);
  double czas_GPU = milliseconds / 1000;

  if ( * wyjscie == 1) {
    cout << "GPU: " << liczba << " to liczba zlozona" << endl;
  } else {
    cout << "GPU" << liczba << " to liczba pierwsza" << endl;
  }

  cout << "Czas na GPU: " << czas_GPU << " s " << endl;

  hipFree(dev_a);

  start = clock();
  czy_pierwsza = pierwsza(liczba);
  end = clock();
  if (czy_pierwsza == 0) {
    cout << "CPU: liczba pierwsza" << endl;
    cout << czy_pierwsza << endl;
  } else {
    cout << "CPU: liczba zlozona" << endl;
    cout << czy_pierwsza << endl;
  }

  float czas_CPU = float(end - start) / float(CLOCKS_PER_SEC);
  cout << "Czas na CPU: " << czas_CPU;
  cout << " s " << endl;

  float przyspieszenie = czas_CPU / czas_GPU;

  cout << "GPU jest " << przyspieszenie << " szybsze niż CPU." << endl;

  return 0;
}